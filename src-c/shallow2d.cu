
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>

//ldoc on
/**
 * ## Implementation
 *
 * The actually work of computing the fluxes and speeds is done
 * by local (`static`) helper functions that take as arguments
 * pointers to all the individual fields.  This is helpful to the
 * compilers, since by specifying the `restrict` keyword, we are
 * promising that we will not access the field data through the
 * wrong pointer.  This lets the compiler do a better job with
 * vectorization.
 */


static const float g = 9.8;

__device__
static
void shallow2dv_flux(float* __restrict__ fh,
                     float* __restrict__ fhu,
                     float* __restrict__ fhv,
                     float* __restrict__ gh,
                     float* __restrict__ ghu,
                     float* __restrict__ ghv,
                     const float* __restrict__ h,
                     const float* __restrict__ hu,
                     const float* __restrict__ hv,
                     float g,
                     int ncell)
{
/*
    		memcpy(fh, hu, ncell * sizeof(float));
    		memcpy(gh, hv, ncell * sizeof(float));
*/

	int indexX = blockIdx.x * blockDim.x + threadIdx.x;
        int cudaStrideX = blockDim.x * gridDim.x;

    for (int i = indexX; i < ncell; i += cudaStrideX) {
   	fh[i] = hu[i];
	gh[i] = hv[i];
        float hi = h[i], hui = hu[i], hvi = hv[i];
        float inv_h = 1/hi;
	fhu[i] = hui*hui*inv_h + (0.5f*g)*hi*hi;
        fhv[i] = hui*hvi*inv_h;
        ghu[i] = hui*hvi*inv_h;
        ghv[i] = hvi*hvi*inv_h + (0.5f*g)*hi*hi;
    }
}

__device__
static
void shallow2dv_speed(float* __restrict__ cxy,
                      const float* __restrict__ h,
                      const float* __restrict__ hu,
                      const float* __restrict__ hv,
                      float g,
                      int ncell)
{
    float cx = cxy[0];
    float cy = cxy[1];
    
	int indexX = blockIdx.x * blockDim.x + threadIdx.x;
	int cudaStrideX = blockDim.x * gridDim.x;

    for (int i = indexX; i < ncell; i += cudaStrideX) {
        float hi = h[i];
        float inv_hi = 1.0f/hi;
        float root_gh = sqrtf(g * hi);
        float cxi = fabsf(hu[i] * inv_hi) + root_gh;
        float cyi = fabsf(hv[i] * inv_hi) + root_gh;
        if (cx < cxi) cx = cxi;
        if (cy < cyi) cy = cyi;
    }
    cxy[0] = cx;
    cxy[1] = cy;
}

__global__
void shallow2d_flux(float* FU, float* GU, const float* U,
                    int ncell, int field_stride)
{
    shallow2dv_flux(FU, FU+field_stride, FU+2*field_stride,
                    GU, GU+field_stride, GU+2*field_stride,
                    U,  U +field_stride, U +2*field_stride,
                    g, ncell);
}

__global__
void shallow2d_speed(float* __restrict__  cxy, const float* __restrict__ U,
                     int ncell, int field_stride)
{
    shallow2dv_speed(cxy, U, U+field_stride, U+2*field_stride, g, ncell);
}
