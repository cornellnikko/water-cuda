#include "stepper.h"
#include "shallow2d.h"
#include "minmod.h"
#include "meshio.h"

#include <string>
#include <cmath>
#include <cstring>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <unistd.h>

#include <thrust/host_vector.h> 
#include <thrust/device_vector.h> 
#include <thrust/copy.h> 
#include <thrust/fill.h> 
#include <thrust/sequence.h>


#ifdef _OPENMP
#include <omp.h>
#elif defined SYSTIME
#include <sys/time.h>
#endif

extern "C" {
#include <lua.h>
#include <lauxlib.h>
#include <lualib.h>
}

//ldoc on
/**
 * # Driver code
 *
 * The driver code is where we put together the time stepper and
 * the physics routines to actually solve the equations and make
 * pretty pictures of the solutions.
 *
 * For the driver, we need to put everything together: we're running
 * a `Central2D` solver for the `Shallow2D` physics with a `MinMod`
 * limiter:
 */

typedef Central2D< Shallow2D, MinMod<Shallow2D::real> > Sim;

/**
 * ## Lua driver routines
 *
 * A better way to manage simulation parameters is by a scripting
 * language.  Python is a popular choice, but I prefer Lua for many
 * things (not least because it is an easy build).  It's also quite
 * cheap to call a Lua function for every point in a mesh
 * (less so for Python, though it probably won't make much difference).
 *
 * ### Lua helpers
 *
 * We want to be able to get numbers and strings with a default value
 * when nothing is specified.  Lua 5.3 has this as a built-in, I think,
 * but the following codes are taken from earlier versions of Lua.
 */

double lget_number(lua_State* L, const char* name, double x)
{
    lua_getfield(L, 1, name);
    if (lua_type(L, -1) != LUA_TNIL) {
        if (lua_type(L, -1) != LUA_TNUMBER)
            luaL_error(L, "Expected %s to be a number", name);
        x = lua_tonumber(L, -1);
    }
    lua_pop(L, 1);
    return x;
}


int lget_int(lua_State* L, const char* name, int x)
{
    lua_getfield(L, 1, name);
    if (lua_type(L, -1) != LUA_TNIL) {
        if (lua_type(L, -1) != LUA_TNUMBER)
            luaL_error(L, "Expected %s to be a number", name);
        x = lua_tointeger(L, -1);
    }
    lua_pop(L, 1);
    return x;
}


const char* lget_string(lua_State* L, const char* name, const char* x)
{
    lua_getfield(L, 1, name);
    if (lua_type(L, -1) != LUA_TNIL) {
        if (lua_type(L, -1) != LUA_TSTRING)
            luaL_error(L, "Expected %s to be a string", name);
        x = lua_tostring(L, -1);
    }
    lua_pop(L, 1);
    return x;
}


/**
 * ### Lua callback functions
 *
 * We specify the initial conditions by providing the simulator
 * with a callback function to be called at each cell center.
 * The callback function is assumed to be the `init` field of
 * a table at index 1.
 */

class LuaInit {
public:
    LuaInit(lua_State* L) : L(L) {}

    void push_init() {
        lua_getfield(L, 1, "init");
        if (lua_type(L, -1) != LUA_TFUNCTION)
            luaL_error(L, "Expected init to be a string");
    }
    
    void operator()(Sim::vec& u, double x, double y) {
        lua_pushvalue(L, -1);
        lua_pushnumber(L, x);
        lua_pushnumber(L, y);
        int nfield = u.size();
        lua_call(L, 2, nfield);
        for (int k = 0; k < nfield; ++k)
            u[k] = lua_tonumber(L, k-nfield);
        lua_pop(L, nfield);
    }
    
private:
    lua_State* L;
};


/**
 * ### Running the simulation
 *
 * The `run_sim` function looks a lot like the main routine of the
 * "ordinary" command line driver.  We specify the initial conditions
 * by providing the simulator with a callback function to be called at
 * each cell center.  Note that we have two different options for
 * timing the steps -- we can use the OpenMP timing routines
 * (preferable if OpenMP is available) or the POSIX `gettimeofday`
 * if the `SYSTIME` macro is defined.  If there's no OpenMP and
 * `SYSTIME` is undefined, we fall back to just printing the number
 * of steps without timing information.
 */

int run_sim(lua_State* L)
{
    int n = lua_gettop(L);
    if (n != 1 || !lua_istable(L, 1))
        luaL_error(L, "Argument must be a table");

    double w = lget_number(L, "w", 2.0);
    double h = lget_number(L, "h", w);
    double cfl = lget_number(L, "cfl", 0.45);
    double ftime = lget_number(L, "ftime", 0.01);
    int nx = lget_int(L, "nx", 200);
    int ny = lget_int(L, "ny", nx);
    int frames = lget_int(L, "frames", 50);
    const char* fname = lget_string(L, "out", "sim.out");

    Sim sim(w, h, nx, ny);
    
    LuaInit initf(L);
    initf.push_init();
    sim.init(initf);
    lua_pop(L,1);
    
    printf("%g %g %d %d %g %d %g\n", w, h, nx, ny, cfl, frames, ftime);
    SimViz<Sim> viz(fname, sim);
    sim.solution_check();
    viz.write_frame();

    double tcompute = 0;
    for (int i = 0; i < frames; ++i) {
#ifdef _OPENMP
        double t0 = omp_get_wtime();
        int nstep = sim.run(ftime);
        double t1 = omp_get_wtime();
        double elapsed = t1-t0;
#elif defined SYSTIME
        struct timeval t0, t1;
        gettimeofday(&t0, NULL);
        int nstep = sim.run(ftime);
        gettimeofday(&t1, NULL);
        double elapsed = (t1.tv_sec-t0.tv_sec) + (t1.tv_usec-t0.tv_usec)*1e-6;
#else
        int nstep = sim.run(ftime);
        double elapsed = 0;
#endif
        sim.solution_check();
        tcompute += elapsed;
        printf("  Time: %e (%e for %d steps)\n", elapsed, elapsed/nstep, nstep);
        viz.write_frame();
    }
    printf("Total compute time: %e\n", tcompute);

    return 0;
}


/**
 * ### Main
 *
 * The main routine has the usage pattern
 *
 *     lshallow tests.lua args
 *
 * where `tests.lua` has a call to the `simulate` function to run
 * the simulation.  The arguments after the Lua file name are passed
 * into the Lua script via a global array called `args`.
 */

int main(int argc, char** argv)
{
    if (argc < 2) {
        fprintf(stderr, "Usage: %s fname args\n", argv[0]);
        return -1;
    }

    lua_State* L = luaL_newstate();
    luaL_openlibs(L);
    lua_register(L, "simulate", run_sim);

    lua_newtable(L);
    for (int i = 2; i < argc; ++i) {
        lua_pushstring(L, argv[i]);
        lua_rawseti(L, 1, i-1);
    }
    lua_setglobal(L, "args");

    if (luaL_dofile(L, argv[1]))
        printf("%s\n", lua_tostring(L,-1));
    lua_close(L);
    return 0;
}
